#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/parametric_res_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void EltwiseMax(const int nthreads, const Dtype* tx1, const Dtype* tx2, Dtype* m) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    m[index] = (tx1[index]>tx2[index]) ? tx1[index] : tx2[index];
  }
}

template <typename Dtype>
void ParametricResLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const Dtype theta = this->blobs_[0]->cpu_data()[0]; // get the parameter
  const Dtype* x1 = bottom[0]->gpu_data();
  const Dtype* x2 = bottom[1]->gpu_data();
  Dtype* tx1 = tx1_.mutable_gpu_data();
  Dtype* tx2 = tx2_.mutable_gpu_data();
  caffe_gpu_memcpy(count*sizeof(Dtype), x1, tx1);
  caffe_gpu_scal(count, theta, tx1); // tx1 <- \theta x1
  caffe_gpu_memcpy(count*sizeof(Dtype), x2, tx2);
  caffe_gpu_scal(count, theta, tx2); // tx2 <- \theta x2
  // find max
  Dtype* m = m_.mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  EltwiseMax<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, tx1, tx2, m);
  caffe_gpu_sub(count, tx1, m, tx1);  // tx1 <- \theta x1 - m
  caffe_gpu_sub(count, tx2, m, tx2);  // tx2 <- \theta x2 - m
  // store the difference in m. for gradient
  caffe_gpu_sub(count, tx1, tx2, m);  // m <- \theta x1 - \theta x2
  // exp
  caffe_gpu_exp(count, tx1, tx1);
  caffe_gpu_exp(count, tx2, tx2);
  // denominator
  Dtype* denom = denom_.mutable_gpu_data();
  caffe_gpu_add(count, tx1, tx2, denom);
  Dtype* y = top[0]->mutable_gpu_data();
  Dtype* buff = buff_.mutable_gpu_data();
  caffe_gpu_mul(count, x1, tx1, buff);
  caffe_gpu_mul(count, x2, tx2, y);
  caffe_gpu_add(count, buff, y, y);
  caffe_gpu_div(count, y, denom, y);
}

template <typename Dtype>
void ParametricResLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // after forward pass,
  const int count = bottom[0]->count();
  const Dtype* x1 = bottom[0]->gpu_data();
  const Dtype* x2 = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* tx1 = tx1_.mutable_gpu_data();
  Dtype* tx2 = tx2_.mutable_gpu_data();
  Dtype* denom = denom_.mutable_gpu_data();
  Dtype* buff = buff_.mutable_gpu_data();
  Dtype* m = m_.mutable_gpu_data(); // after forward pass m <- \theta x1 - \theta x2
  // need denominator squared
  caffe_gpu_mul(count, denom, denom, denom);
  caffe_gpu_mul(count, tx1, tx2, buff); // buff <- exp( \theta x1 + \theta x2 - 2m )
  if (propagate_down[0]) {
    // gradient w.r.t x1
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_add_scalar(count, Dtype(1), m);
    caffe_gpu_mul(count, m, buff, bottom_diff);
    caffe_gpu_mul(count, tx1, tx1, tx1);
    caffe_gpu_add(count, bottom_diff, tx1, bottom_diff);
    caffe_gpu_div(count, bottom_diff, denom, bottom_diff);
    // finally take into account the top diff
    caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
  }
  if (propagate_down[1]) {
    // gradient w.r.t x1
    Dtype* bottom_diff = bottom[1]->mutable_gpu_diff();
    caffe_gpu_scal(count, Dtype(-1), m);
    caffe_gpu_add_scalar(count, Dtype(2), m); // from tx1-tx2+1 --> tx2-tx1+1
    caffe_gpu_mul(count, m, buff, bottom_diff);
    caffe_gpu_mul(count, tx2, tx2, tx2);
    caffe_gpu_add(count, bottom_diff, tx2, bottom_diff);
    caffe_gpu_div(count, bottom_diff, denom, bottom_diff);
    // finally take into account the top diff
    caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
  }
  // gradient w.r.t theta
  caffe_gpu_div(count, buff, denom, buff);  // buff <- exp( \theta x1 + \theta x2 - 2m ) / ()^2
  caffe_gpu_sub(count, x1, x2, m);
  caffe_gpu_mul(count, m, m, m);  // m <- (x1-x2)^2
  caffe_gpu_mul(count, m, top_diff, m); // take into account the top diff
  Dtype* theta_diff = this->blobs_[0]->mutable_cpu_diff(); // not gpu_diff here, gpu_dot pushes result to host apparently
  caffe_gpu_dot(count, m, buff, theta_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(ParametricResLayer);

} // namespace caffe

